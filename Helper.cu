#include "hip/hip_runtime.h"
// Kaustav Vats (2016048)
// Anubhav Jaiswal (2016014)
// Arshdeep Singh (2016018)

#include <iostream>
#include <cstring>
#include "Helper.h"

using namespace std;

/*
 * Notes:
 * 1. 0 is denoted as blank tile.
 */

/* 
 * Function to print Content of the Structure Node 
 */
void toString(Node * node) {
    cout << "UID:\t" << node->UID << endl;
    cout << "DistanceTop:\t" << node->DT << endl;
    cout << "HeuristicD:\t" << node->HD << endl;
    cout << "Data:\t" << endl;
    for (int i=0; i<N; i++) {
        for (int j=0; j<N; j++) {
            cout << node->Data[i][j] << " ";
        }
        cout << endl;
    }
    if (node->Link != NULL) {
        for (int i=0; i<4; i++) {
            if (node->Link[i].DT != 0) {
                cout << "[" << i << "]\t" << node->Link[i].UID << endl;
            }
        }
    }   
}

/* 
 * Fill all required values of the struct
 */
void Fill(Node * node, int dt, int hd, int data[N][N], Node * link) {
    node->DT = dt;
    node->HD = -1;
    for(int i=0; i<N; i++) {
        for(int j=0; j<N; j++) {
            node->Data[i][j] = data[i][j];
            node->UID[2*(N*i + j)] = data[i][j] + '0';
            node->UID[2*(N*i + j)+1] = ' ';
        }
    }
    node->UID[2*N*N] = '\0';
    node->Link = link;
}

/*
 * DeepcopyData: Copy Data variable in same size array.
 * node: Original Node.
 * copy: Copy data.
 */
void DeepcopyData(Node * node, int copy[N][N]) {
    for (int i=0; i<N; i++) {
        for (int j=0; j<N; j++) {
            copy[i][j] = node->Data[i][j];
        }
    }
}

int getIndexInState(int pos, int state[N][N]) {
    for(int i=0; i<N; i++) {
        for(int j=0; j<N; j++) {
            if(state[i][j] == pos) {
                return i*N + j;
            }
        }
    }
    return -1;
}

/*
 * node: Node from which the heuristic distance needs to be calculated
 * finalState: the final state array of the problem
 * Returns: The heuristic distance from the givn node state to the final state
 */
void UpdateHD(Node& node, int finalState[N][N]) {
    if(node.HD > -1) {
        return;
    }

    // for(int i=0; i<N; i++) {
    //     for(int j=0; j<N; j++) {
    //         cout << node.Data[i][j] << " ";
    //     }
    //     cout << endl;
    // }

    int nodeArr[N*N], finalArr[N*N];
    for(int i=0; i<N; i++) {
        for(int j=0; j<N; j++) {
            int pos = i*N + j;
            // cout << pos << " " << node.Data[i][j] << endl;
            nodeArr[node.Data[i][j]] = pos;
            finalArr[finalState[i][j]] = pos;
            // cout << node.Data[i][j] << " " << pos << endl;
            // int b = getIndexInState(pos, node.Data);
            // int g = getIndexInState(pos, finalState);
            // sum += abs(b % N - g % N);
            // sum += abs(b / N - g / N);
        }
    }

    int HDvalue = 0;
    for(int i=1; i<N*N; i++){
        int b = nodeArr[i];
        int g = finalArr[i];
        HDvalue += abs(b % N - g % N);
        HDvalue += abs(b / N - g / N);
    }
    node.HD = HDvalue;
}

/*
 * This function empty tile in the node.
 * Returns: Add X, Y coordinate of 0 Tile in Array 'arr'
 */
void FindZeros(int data[N][N], int * x, int * y) {
    for(int i=0; i<N; i++) {
        for(int j=0; j<N; j++) {
            if (data[i][j] == 0) {
                *x = i;
                *y = j;
                return;
            }
        }
    }
}

/*
 * Helper function to print 2D Array
 */
void temp_display(int data[N][N]) {
    for (int i=0; i<N; i++) {
        for (int j=0; j<N; j++) {
            cout << data[i][j] << " ";
        }
        cout << endl;
    }
}

__global__ void findZero(int * d_data, unsigned int * d_xy) {
    // printf("%d\t", d_data[threadIdx.x]);
    if (d_data[threadIdx.x] == 0 && threadIdx.x < N*N) {
        d_xy[0] = threadIdx.x;
    }
}


/* 
 * Find all neighbours of the Node,
 * Next possible state of the node.
 * Case 1: Corners of the Grid.
 * Case 2: When 0 is on the Edge of the grid
 * Case 3: When 0 in anywhere except above above possible cases
 * Return: Add all Neighbours in 2nd Argument
 */
 void GetNeighbours(Node * currentNode) {
    Node * neighbours;
    neighbours = (Node *) malloc(sizeof(Node) * 4);
    int * d_data;
    unsigned int * d_xy;
    hipMalloc((void**) &d_data, sizeof(int)*N*N);
    hipMalloc((void**) &d_xy, sizeof(unsigned int));
    hipMemcpy(d_data, currentNode.Data, sizeof(int)*N*N , hipMemcpyHostToDevice);
    
    findZero<<<1, 32>>>(d_data, d_xy); 
    unsigned int * xy;
    xy = (unsigned int *) malloc(sizeof(unsigned int)); 

    hipMemcpy(xy, d_xy, sizeof(unsigned int), hipMemcpyDeviceToHost);
    // cout << "xy: " << xy[0] << endl;
    int x = (int) floor(xy[0] / N);
    int y = (int)xy[0] - x*N;
    // cout << "x: " << x << " y: " << y << endl;
    
    // Case 1: Corner 1, 2, 3, 4
    if (x == 0 && y == 0) {
        int Temp[N][N];
        DeepcopyData(currentNode, Temp);
        Temp[x][y] = Temp[x][y+1];
        Temp[x][y+1] = 0;
        Node n1;
        Fill(&n1, currentNode->DT+1, -1, Temp, NULL);
        Temp[x][y+1] = Temp[x][y];
        Temp[x][y] = 0;
        neighbours[0] = n1;

        Temp[x][y] = Temp[x+1][y];
        Temp[x+1][y] = 0;
        Node n2;
        Fill(&n2, currentNode->DT+1, -1, Temp, NULL);
        Temp[x+1][y] = Temp[x][y];
        Temp[x][y] = 0;
        neighbours[1] = n2;
        neighbours[2] = {0};
        neighbours[3] = {0};
        currentNode->Link = neighbours;
        // toString(currentNode);
    }
    else if (x == 0 && y == N-1) {
        // Need to repeat same for rest of the conditions
        int Temp[N][N];
        DeepcopyData(currentNode, Temp);
        Temp[x][y] = Temp[x][y-1];
        Temp[x][y-1] = 0;
        Node n1;
        Fill(&n1, currentNode->DT+1, -1, Temp, NULL);
        Temp[x][y-1] = Temp[x][y];
        Temp[x][y] = 0;
        neighbours[0] = n1;

        Temp[x][y] = Temp[x+1][y];
        Temp[x+1][y] = 0;
        Node n2;
        Fill(&n2, currentNode->DT+1, -1, Temp, NULL);
        Temp[x+1][y] = Temp[x][y];
        Temp[x][y] = 0;
        neighbours[1] = n2;
        neighbours[2] = {0};
        neighbours[3] = {0};
        currentNode->Link = neighbours;
        // toString(currentNode);
    }
    else if (x == N-1 && y == 0) {
        int Temp[N][N];
        DeepcopyData(currentNode, Temp);
        Temp[x][y] = Temp[x][y+1];
        Temp[x][y+1] = 0;
        Node n1;
        Fill(&n1, currentNode->DT+1, -1, Temp, NULL);
        Temp[x][y+1] = Temp[x][y];
        Temp[x][y] = 0;
        neighbours[0] = n1;

        Temp[x][y] = Temp[x-1][y];
        Temp[x-1][y] = 0;
        Node n2;
        Fill(&n2, currentNode->DT+1, -1, Temp, NULL);
        Temp[x-1][y] = Temp[x][y];
        Temp[x][y] = 0;
        neighbours[1] = n2;
        neighbours[2] = {0};
        neighbours[3] = {0};
        currentNode->Link = neighbours;
        // toString(currentNode);
    }
    else if (x == N-1 && y == N-1) {
        int Temp[N][N];
        DeepcopyData(currentNode, Temp);
        Temp[x][y] = Temp[x][y-1];
        Temp[x][y-1] = 0;
        Node n1;
        Fill(&n1, currentNode->DT+1, -1, Temp, NULL);
        Temp[x][y-1] = Temp[x][y];
        Temp[x][y] = 0;
        neighbours[0] = n1;

        Temp[x][y] = Temp[x-1][y];
        Temp[x-1][y] = 0;
        Node n2;
        Fill(&n2, currentNode->DT+1, -1, Temp, NULL);
        Temp[x-1][y] = Temp[x][y];
        Temp[x][y] = 0;
        neighbours[1] = n2;
        neighbours[2] = {0};
        neighbours[3] = {0};
        currentNode->Link = neighbours;
        // toString(currentNode);
    }
    // Case 2: Edge 1, 2, 3, 4
    else if (x == 0) {
        int Temp[N][N];
        DeepcopyData(currentNode, Temp);
        Temp[x][y] = Temp[x][y+1];
        Temp[x][y+1] = 0;
        Node n1;
        Fill(&n1, currentNode->DT+1, -1, Temp, NULL);
        Temp[x][y+1] = Temp[x][y];
        Temp[x][y] = 0;
        neighbours[0] = n1;

        Temp[x][y] = Temp[x][y-1];
        Temp[x][y-1] = 0;
        Node n2;
        Fill(&n2, currentNode->DT+1, -1, Temp, NULL);
        Temp[x][y-1] = Temp[x][y];
        Temp[x][y] = 0;
        neighbours[1] = n2;

        Temp[x][y] = Temp[x+1][y];
        Temp[x+1][y] = 0;
        Node n3;
        Fill(&n3, currentNode->DT+1, -1, Temp, NULL);
        Temp[x+1][y] = Temp[x][y];
        Temp[x][y] = 0;
        neighbours[2] = n3;
        neighbours[3] = {0};    
        currentNode->Link = neighbours;
        // toString(currentNode);

    }
    else if (y == 0) {
        int Temp[N][N];
        DeepcopyData(currentNode, Temp);
        Temp[x][y] = Temp[x][y+1];
        Temp[x][y+1] = 0;
        Node n1;
        Fill(&n1, currentNode->DT+1, -1, Temp, NULL);
        Temp[x][y+1] = Temp[x][y];
        Temp[x][y] = 0;
        neighbours[0] = n1;

        Temp[x][y] = Temp[x-1][y];
        Temp[x-1][y] = 0;
        Node n2;
        Fill(&n2, currentNode->DT+1, -1, Temp, NULL);
        Temp[x-1][y] = Temp[x][y];
        Temp[x][y] = 0;
        neighbours[1] = n2;

        Temp[x][y] = Temp[x+1][y];
        Temp[x+1][y] = 0;
        Node n3;
        Fill(&n3, currentNode->DT+1, -1, Temp, NULL);
        Temp[x+1][y] = Temp[x][y];
        Temp[x][y] = 0;
        neighbours[2] = n3;
        neighbours[3] = {0};    
        currentNode->Link = neighbours;
        // toString(currentNode);

    }
    else if (x == N-1) {
        int Temp[N][N];
        DeepcopyData(currentNode, Temp);
        Temp[x][y] = Temp[x][y+1];
        Temp[x][y+1] = 0;
        Node n1;
        Fill(&n1, currentNode->DT+1, -1, Temp, NULL);
        Temp[x][y+1] = Temp[x][y];
        Temp[x][y] = 0;
        neighbours[0] = n1;

        Temp[x][y] = Temp[x][y-1];
        Temp[x][y-1] = 0;
        Node n2;
        Fill(&n2, currentNode->DT+1, -1, Temp, NULL);
        Temp[x][y-1] = Temp[x][y];
        Temp[x][y] = 0;
        neighbours[1] = n2;

        Temp[x][y] = Temp[x-1][y];
        Temp[x-1][y] = 0;
        Node n3;
        Fill(&n3, currentNode->DT+1, -1, Temp, NULL);
        Temp[x-1][y] = Temp[x][y];
        Temp[x][y] = 0;
        neighbours[2] = n3;
        neighbours[3] = {0};
        currentNode->Link = neighbours;
        // toString(currentNode);
    }
    else if (y == N-1) {
        int Temp[N][N];
        DeepcopyData(currentNode, Temp);
        Temp[x][y] = Temp[x+1][y];
        Temp[x+1][y] = 0;
        Node n1;
        Fill(&n1, currentNode->DT+1, -1, Temp, NULL);
        Temp[x+1][y] = Temp[x][y];
        Temp[x][y] = 0;
        neighbours[0] = n1;

        Temp[x][y] = Temp[x][y-1];
        Temp[x][y-1] = 0;
        Node n2;
        Fill(&n2, currentNode->DT+1, -1, Temp, NULL);
        Temp[x][y-1] = Temp[x][y];
        Temp[x][y] = 0;
        neighbours[1] = n2;

        Temp[x][y] = Temp[x-1][y];
        Temp[x-1][y] = 0;
        Node n3;
        Fill(&n3, currentNode->DT+1, -1, Temp, NULL);
        Temp[x-1][y] = Temp[x][y];
        Temp[x][y] = 0;
        neighbours[2] = n3;
        neighbours[3] = {0};
        currentNode->Link = neighbours;
        // toString(currentNode);
    }
    // Case 3: Tile 0 Anywhere, except above possible location.
    else {
        int Temp[N][N];
        DeepcopyData(currentNode, Temp);
        Temp[x][y] = Temp[x+1][y];
        Temp[x+1][y] = 0;
        Node n1;
        Fill(&n1, currentNode->DT+1, -1, Temp, NULL);
        Temp[x+1][y] = Temp[x][y];
        Temp[x][y] = 0;
        neighbours[0] = n1;

        Temp[x][y] = Temp[x][y-1];
        Temp[x][y-1] = 0;
        Node n2;
        Fill(&n2, currentNode->DT+1, -1, Temp, NULL);
        Temp[x][y-1] = Temp[x][y];
        Temp[x][y] = 0;
        neighbours[1] = n2;

        Temp[x][y] = Temp[x-1][y];
        Temp[x-1][y] = 0;
        Node n3;
        Fill(&n3, currentNode->DT+1, -1, Temp, NULL);
        Temp[x-1][y] = Temp[x][y];
        Temp[x][y] = 0;
        neighbours[2] = n3;

        Temp[x][y] = Temp[x][y+1];
        Temp[x][y+1] = 0;
        Node n4;
        Fill(&n4, currentNode->DT+1, -1, Temp, NULL);
        Temp[x][y+1] = Temp[x][y];
        Temp[x][y] = 0;
        neighbours[3] = n4;
        currentNode->Link = neighbours;
        // toString(currentNode);
    }

    return;
}

/* 
 * Find all neighbours of the Node,
 * Next possible state of the node.
 * Case 1: Corners of the Grid.
 * Case 2: When 0 is on the Edge of the grid
 * Case 3: When 0 in anywhere except above above possible cases
 * Return: Add all Neighbours in 2nd Argument
 */
void GetNeighbours(Node * currentNode) {
    Node * neighbours;
    neighbours = (Node *) malloc(sizeof(Node) * 4);
    int x, y;
    FindZeros(currentNode->Data, &x, &y);
    // cout << "Zero Coordinates\nX: " << x << "\tY: " << y << endl;

    // Case 1: Corner 1, 2, 3, 4
    if (x == 0 && y == 0) {
        int Temp[N][N];
        DeepcopyData(currentNode, Temp);
        Temp[x][y] = Temp[x][y+1];
        Temp[x][y+1] = 0;
        Node n1;
        Fill(&n1, currentNode->DT+1, -1, Temp, NULL);
        Temp[x][y+1] = Temp[x][y];
        Temp[x][y] = 0;
        neighbours[0] = n1;

        Temp[x][y] = Temp[x+1][y];
        Temp[x+1][y] = 0;
        Node n2;
        Fill(&n2, currentNode->DT+1, -1, Temp, NULL);
        Temp[x+1][y] = Temp[x][y];
        Temp[x][y] = 0;
        neighbours[1] = n2;
        neighbours[2] = {0};
        neighbours[3] = {0};
        currentNode->Link = neighbours;
        // toString(currentNode);
    }
    else if (x == 0 && y == N-1) {
        // Need to repeat same for rest of the conditions
        int Temp[N][N];
        DeepcopyData(currentNode, Temp);
        Temp[x][y] = Temp[x][y-1];
        Temp[x][y-1] = 0;
        Node n1;
        Fill(&n1, currentNode->DT+1, -1, Temp, NULL);
        Temp[x][y-1] = Temp[x][y];
        Temp[x][y] = 0;
        neighbours[0] = n1;

        Temp[x][y] = Temp[x+1][y];
        Temp[x+1][y] = 0;
        Node n2;
        Fill(&n2, currentNode->DT+1, -1, Temp, NULL);
        Temp[x+1][y] = Temp[x][y];
        Temp[x][y] = 0;
        neighbours[1] = n2;
        neighbours[2] = {0};
        neighbours[3] = {0};
        currentNode->Link = neighbours;
        // toString(currentNode);
    }
    else if (x == N-1 && y == 0) {
        int Temp[N][N];
        DeepcopyData(currentNode, Temp);
        Temp[x][y] = Temp[x][y+1];
        Temp[x][y+1] = 0;
        Node n1;
        Fill(&n1, currentNode->DT+1, -1, Temp, NULL);
        Temp[x][y+1] = Temp[x][y];
        Temp[x][y] = 0;
        neighbours[0] = n1;

        Temp[x][y] = Temp[x-1][y];
        Temp[x-1][y] = 0;
        Node n2;
        Fill(&n2, currentNode->DT+1, -1, Temp, NULL);
        Temp[x-1][y] = Temp[x][y];
        Temp[x][y] = 0;
        neighbours[1] = n2;
        neighbours[2] = {0};
        neighbours[3] = {0};
        currentNode->Link = neighbours;
        // toString(currentNode);
    }
    else if (x == N-1 && y == N-1) {
        int Temp[N][N];
        DeepcopyData(currentNode, Temp);
        Temp[x][y] = Temp[x][y-1];
        Temp[x][y-1] = 0;
        Node n1;
        Fill(&n1, currentNode->DT+1, -1, Temp, NULL);
        Temp[x][y-1] = Temp[x][y];
        Temp[x][y] = 0;
        neighbours[0] = n1;

        Temp[x][y] = Temp[x-1][y];
        Temp[x-1][y] = 0;
        Node n2;
        Fill(&n2, currentNode->DT+1, -1, Temp, NULL);
        Temp[x-1][y] = Temp[x][y];
        Temp[x][y] = 0;
        neighbours[1] = n2;
        neighbours[2] = {0};
        neighbours[3] = {0};
        currentNode->Link = neighbours;
        // toString(currentNode);
    }
    // Case 2: Edge 1, 2, 3, 4
    else if (x == 0) {
        int Temp[N][N];
        DeepcopyData(currentNode, Temp);
        Temp[x][y] = Temp[x][y+1];
        Temp[x][y+1] = 0;
        Node n1;
        Fill(&n1, currentNode->DT+1, -1, Temp, NULL);
        Temp[x][y+1] = Temp[x][y];
        Temp[x][y] = 0;
        neighbours[0] = n1;

        Temp[x][y] = Temp[x][y-1];
        Temp[x][y-1] = 0;
        Node n2;
        Fill(&n2, currentNode->DT+1, -1, Temp, NULL);
        Temp[x][y-1] = Temp[x][y];
        Temp[x][y] = 0;
        neighbours[1] = n2;

        Temp[x][y] = Temp[x+1][y];
        Temp[x+1][y] = 0;
        Node n3;
        Fill(&n3, currentNode->DT+1, -1, Temp, NULL);
        Temp[x+1][y] = Temp[x][y];
        Temp[x][y] = 0;
        neighbours[2] = n3;
        neighbours[3] = {0};    
        currentNode->Link = neighbours;
        // toString(currentNode);

    }
    else if (y == 0) {
        int Temp[N][N];
        DeepcopyData(currentNode, Temp);
        Temp[x][y] = Temp[x][y+1];
        Temp[x][y+1] = 0;
        Node n1;
        Fill(&n1, currentNode->DT+1, -1, Temp, NULL);
        Temp[x][y+1] = Temp[x][y];
        Temp[x][y] = 0;
        neighbours[0] = n1;

        Temp[x][y] = Temp[x-1][y];
        Temp[x-1][y] = 0;
        Node n2;
        Fill(&n2, currentNode->DT+1, -1, Temp, NULL);
        Temp[x-1][y] = Temp[x][y];
        Temp[x][y] = 0;
        neighbours[1] = n2;

        Temp[x][y] = Temp[x+1][y];
        Temp[x+1][y] = 0;
        Node n3;
        Fill(&n3, currentNode->DT+1, -1, Temp, NULL);
        Temp[x+1][y] = Temp[x][y];
        Temp[x][y] = 0;
        neighbours[2] = n3;
        neighbours[3] = {0};    
        currentNode->Link = neighbours;
        // toString(currentNode);

    }
    else if (x == N-1) {
        int Temp[N][N];
        DeepcopyData(currentNode, Temp);
        Temp[x][y] = Temp[x][y+1];
        Temp[x][y+1] = 0;
        Node n1;
        Fill(&n1, currentNode->DT+1, -1, Temp, NULL);
        Temp[x][y+1] = Temp[x][y];
        Temp[x][y] = 0;
        neighbours[0] = n1;

        Temp[x][y] = Temp[x][y-1];
        Temp[x][y-1] = 0;
        Node n2;
        Fill(&n2, currentNode->DT+1, -1, Temp, NULL);
        Temp[x][y-1] = Temp[x][y];
        Temp[x][y] = 0;
        neighbours[1] = n2;

        Temp[x][y] = Temp[x-1][y];
        Temp[x-1][y] = 0;
        Node n3;
        Fill(&n3, currentNode->DT+1, -1, Temp, NULL);
        Temp[x-1][y] = Temp[x][y];
        Temp[x][y] = 0;
        neighbours[2] = n3;
        neighbours[3] = {0};
        currentNode->Link = neighbours;
        // toString(currentNode);
    }
    else if (y == N-1) {
        int Temp[N][N];
        DeepcopyData(currentNode, Temp);
        Temp[x][y] = Temp[x+1][y];
        Temp[x+1][y] = 0;
        Node n1;
        Fill(&n1, currentNode->DT+1, -1, Temp, NULL);
        Temp[x+1][y] = Temp[x][y];
        Temp[x][y] = 0;
        neighbours[0] = n1;

        Temp[x][y] = Temp[x][y-1];
        Temp[x][y-1] = 0;
        Node n2;
        Fill(&n2, currentNode->DT+1, -1, Temp, NULL);
        Temp[x][y-1] = Temp[x][y];
        Temp[x][y] = 0;
        neighbours[1] = n2;

        Temp[x][y] = Temp[x-1][y];
        Temp[x-1][y] = 0;
        Node n3;
        Fill(&n3, currentNode->DT+1, -1, Temp, NULL);
        Temp[x-1][y] = Temp[x][y];
        Temp[x][y] = 0;
        neighbours[2] = n3;
        neighbours[3] = {0};
        currentNode->Link = neighbours;
        // toString(currentNode);
    }
    // Case 3: Tile 0 Anywhere, except above possible location.
    else {
        int Temp[N][N];
        DeepcopyData(currentNode, Temp);
        Temp[x][y] = Temp[x+1][y];
        Temp[x+1][y] = 0;
        Node n1;
        Fill(&n1, currentNode->DT+1, -1, Temp, NULL);
        Temp[x+1][y] = Temp[x][y];
        Temp[x][y] = 0;
        neighbours[0] = n1;

        Temp[x][y] = Temp[x][y-1];
        Temp[x][y-1] = 0;
        Node n2;
        Fill(&n2, currentNode->DT+1, -1, Temp, NULL);
        Temp[x][y-1] = Temp[x][y];
        Temp[x][y] = 0;
        neighbours[1] = n2;

        Temp[x][y] = Temp[x-1][y];
        Temp[x-1][y] = 0;
        Node n3;
        Fill(&n3, currentNode->DT+1, -1, Temp, NULL);
        Temp[x-1][y] = Temp[x][y];
        Temp[x][y] = 0;
        neighbours[2] = n3;

        Temp[x][y] = Temp[x][y+1];
        Temp[x][y+1] = 0;
        Node n4;
        Fill(&n4, currentNode->DT+1, -1, Temp, NULL);
        Temp[x][y+1] = Temp[x][y];
        Temp[x][y] = 0;
        neighbours[3] = n4;
        currentNode->Link = neighbours;
        // toString(currentNode);
    }

    return;
}

/*
 * This function check if current state is the final state or not.
 * Returns: 1 if states does not match, On correct match returns 0
 */
int checkSolution(Node * node, int FinalState[N][N]) {
    for (int i=0; i<N; i++) {
        for (int j=0; j<N; j++) {
            if (node->Data[i][j] != FinalState[i][j]) {
                return 1;
            }
        }
    }
    return 0;
}
