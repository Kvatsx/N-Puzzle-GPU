#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <bits/stdc++.h>
#include "Helper.h"
using namespace std;

#define BLOCK_SIZE 512
#define CUDA_FUNC __host__ __device__
#define MAX_QUEUE_SIZE 1000

class PriorityQueue {
private:

	Node heapArr[MAX_QUEUE_SIZE];
	unsigned int length;

public:

	CUDA_FUNC PriorityQueue() {
		this->length = 0;
	}


	CUDA_FUNC int compareNode(int a, int b) {

		return this->compareNode(this->heapArr[a], this->heapArr[b]);

	}

	CUDA_FUNC int compareNode(Node &left, Node &right) {
		int leftSum = left.HD + left.DT;
        int rightSum = right.HD + right.DT;
        return leftSum < rightSum;
	}

	CUDA_FUNC void swap(int a, int b) {
		Node temp = this->heapArr[a];
		this->heapArr[a] = this->heapArr[b];
		this->heapArr[b] = temp;
	}

	CUDA_FUNC void increase_val(int i, Node val) {
		// printf("asasdsa %d %d %d %d\n", i, val, this->heapArr[i], val < this->heapArr[i]);
		if(this->heapArr[i].DT != -1 && this->compareNode(val, this->heapArr[i]))
			return;
		this->heapArr[i] = val;
		// printf("insertion %d", this->heapArr[i]);
		while(i>1 && this->compareNode(i/2, i))
		{
			this->swap(i/2, i);
			i = i/2;
		}
	}

	CUDA_FUNC void insert(Node val) {
		Node NULL_NODE;
		NULL_NODE.DT = -1;
		this->length = this->length + 1;
		this->heapArr[this->length] = NULL_NODE;
		// printf("sdsfsdfdsfsd\n");
		this->increase_val(this->length, val);
		// printf("sdsfsdfdsfsd\n");

	}

	CUDA_FUNC void max_heapify(int i) {
		int left = 2*i;
		int right = 2*i+1;
		int largest = 1;
		if(left <= this->length && ! this->compareNode(left, i))
			largest = left;
		else
			largest = i;
		if(right <= this->length && ! this->compareNode(right, largest))
		    largest = right;

		if(largest != i)
		{
		    this->swap(i, largest);
		    this->max_heapify(largest);
		} 
	 }

	CUDA_FUNC Node top()
	{
		if(this->length == 0) {
			Node NULL_NODE;
			NULL_NODE.DT = -1;
			return NULL_NODE;
		}

		Node max = this->heapArr[1];

		this->heapArr[1] = this->heapArr[this->length];
		this->length--;
		max_heapify(1);
		// for(int i=0; i<this->length; i++) {
		// 	printf("%d", this->heapArr[i]);
		// }

		return max;
	}

	CUDA_FUNC int getLength() {
		return this->length;
	}

	// CUDA_FUNC void printHeap() {
	// 	for(int i=0; i<this->length; i++) {
	// 		printf("%d,%d\t", i, this->heapArr[i]);
	// 	}
	// 	printf("\n");
	// }


};

#define K 10

__global__ void parAStar(PriorityQueue *pqC) {

	int i = blockIdx.x * blockDim.x + threadIdx.x;
	// i=i+1;
	if(i < K) {
		Node val = pqC[i].top();
		printf("%d %d %d\n", i, val.DT, val.HD);
		__syncthreads();
		val = pqC[i].top();
		printf("%d %d %d\n", i, val.DT, val.HD);
		// printf("%d %d\n", i, pqC[i].top());
	}
}

int main() {

	int k = K;
	int blocks = (k / BLOCK_SIZE) + 1;
	printf("%d %d\n", blocks, BLOCK_SIZE);

	PriorityQueue pq[k];

	printf("%lu\n", sizeof(PriorityQueue));
	// printf("sdfsddsfsd");
	for(int i=0; i<k; i++) {
		pq[i] = PriorityQueue();
		for(int j=0; j<10; j++) {
			Node val;
			val.DT = i*k + j;
			val.HD = i*k + j;
			pq[i].insert(val);
		}
	}

	// for(int i=0; i<k; i++) {
	// 	pq[i].printHeap();
	// 	// printf("%d %d\n", i, pq[i].top());
	// }
	

	hipEvent_t start_kernel, stop_kernel;
	hipEventCreate(&start_kernel);
	hipEventCreate(&stop_kernel);
	hipEventRecord(start_kernel);
	// int arr[5] = {1,2,3,4,5};
	PriorityQueue *pqC;
	hipMalloc((void**)&pqC, sizeof(PriorityQueue)*k);
	hipMemcpy(pqC, pq, sizeof(PriorityQueue)*k, hipMemcpyHostToDevice);
	parAStar<<<1, 32>>> (pqC);
	hipMemcpy(pq, pqC, sizeof(PriorityQueue)*k, hipMemcpyDeviceToHost);
	hipEventRecord(stop_kernel);
	hipEventSynchronize(stop_kernel);
	// hipEventElapsedTime(&totalTime, start_kernel, stop_kernel);


	return 0;
}