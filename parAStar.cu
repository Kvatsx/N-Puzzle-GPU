#include <hip/hip_runtime.h>

#include <stdio.h>

#define BLOCK_SIZE 512

__global__ void parAStar(int *Va) {

	int i = blockIdx.x * blockDim.x + threadIdx.x;
	// i=i+1;
	printf("%d\n", Va[i]);
}

int main() {

	int k = 1000;
	int blocks = (k / BLOCK_SIZE) + 1;
	printf("%d %d\n", blocks, BLOCK_SIZE);
	
	// cudaEvent_t start_kernel, stop_kernel;
	// cudaEventCreate(&start_kernel);
	// cudaEventCreate(&stop_kernel);
	// cudaEventRecord(start_kernel);
	int arr[5] = {1,2,3,4,5};
	int* Va;
	hipMalloc((void**)&Va, sizeof(int)*5);
	hipMemcpy(Va, arr, sizeof(int)*5, hipMemcpyHostToDevice);
	parAStar<<<1, 32>>> (Va);
	hipMemcpy(arr, Va, sizeof(int)*5, hipMemcpyDeviceToHost);
	// cudaEventRecord(stop_kernel);
	// cudaEventSynchronize(stop_kernel);
	// cudaEventElapsedTime(&totalTime, start_kernel, stop_kernel);


	return 0;
}