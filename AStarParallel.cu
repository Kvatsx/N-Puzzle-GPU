#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <bits/stdc++.h>
#include "newHelper.h"
using namespace std;

#define FINAL_STATE {{0,1,2}, {3,4,5}, {6,7,8}}
// #define FINAL_STATE {{1,2,3,4}, {5,6,7,8}, {9,10,11,12}, {13,14,15,0}}
#define BLOCK_SIZE 512
#define MAX_QUEUE_SIZE 1000

class PriorityQueue {
private:

	Node heapArr[MAX_QUEUE_SIZE];
	unsigned int length;

public:

	CUDA_FUNC PriorityQueue() {
		this->length = 0;
	}


	CUDA_FUNC int compareNode(int a, int b) {

		return this->compareNode1(this->heapArr[a], this->heapArr[b]);

	}

	CUDA_FUNC int compareNode1(Node &left, Node &right) {

        UpdateHD(left);
        UpdateHD(right);
        // printf("left - uid %s hd %d dt %d\n", left.UID, left.HD, left.DT);
        // printf("right - uid %s hd %d dt %d\n", right.UID, right.HD, right.DT);

		int leftSum = left.HD + left.DT;
        int rightSum = right.HD + right.DT;
        // printf("comp %d\n", leftSum < rightSum);
        if(leftSum == rightSum)
            return left.HD > right.HD;
        return leftSum > rightSum;
	}

	CUDA_FUNC void swap(int a, int b) {
		Node temp = this->heapArr[a];
		this->heapArr[a] = this->heapArr[b];
		this->heapArr[b] = temp;
	}

	CUDA_FUNC void increase_val(int i, Node val) {
		// printf("asasdsa %d %d %d %d\n", i, val, this->heapArr[i], val < this->heapArr[i]);
		// if(this->heapArr[i].DT != -1 && this->compareNode1(val, this->heapArr[i]))
		// 	return;
        // printf("sdfkhsdlfkjadshlfkajdhlfkjdsh %d %d\n", i, this->length);

		this->heapArr[i] = val;
                // printf("VAL%d %d\n", val.HD, i);
		// printf("insertion %d", this->heapArr[i]);
		while(i>1 && this->compareNode(i/2, i))
		{
			this->swap(i/2, i);
			i = i/2;
		}
	}

	CUDA_FUNC void insert(Node val) {
		// Node NULL_NODE;
		// NULL_NODE.DT = -1;
		this->length = this->length + 1;
		// this->heapArr[this->length] = NULL_NODE;
		// printf("sdsfsdfdsfsd\n");
		this->increase_val(this->length, val);
		// printf("sdsfsdfdsfsd\n");
        UpdateHD(val);
        // printf("top of queue %s hd %d dt %d\n", this->heapArr[1].UID, this->heapArr[1].HD, this->heapArr[1].DT);
        // printf("current node %s hd %d dt %d\n", val.UID, val.HD, val.DT);

	}

    

	CUDA_FUNC void max_heapify(int i) {
        while(true) {

    		int left = 2*i;
    		int right = 2*i+1;
    		int largest = 1;
            // printf("dsdfgetwertrwet\n");
    		if(left <= this->length && ! this->compareNode(left, i))
    			largest = left;
    		else
    			largest = i;
    		if(right <= this->length && ! this->compareNode(right, largest))
    		    largest = right;

    		if(largest != i) {
    		    this->swap(i, largest);
    		    i = largest;
    		}
            else {
                break;
            }
        }
	}

	CUDA_FUNC Node top()
	{

		if(this->length == 0) {
			Node NULL_NODE;
			NULL_NODE.DT = -1;
			return NULL_NODE;
		}

		Node max = this->heapArr[1];

		this->heapArr[1] = this->heapArr[this->length];

		this->length--;

		this->max_heapify(1);
		// for(int i=0; i<this->length; i++) {
		// 	printf("%d", this->heapArr[i]);
		// }
        
		return max;
	}

	CUDA_FUNC int getLength() {
		return this->length;
	}

	CUDA_FUNC void printHeap() {
		for(int i=0; i<this->length; i++) {
			printf("(%d : %s)\n", i, this->heapArr[i].UID);
		}
		printf("\n");
	}


};

/*
* Updating the HD of the passed Node
*/
CUDA_FUNC void UpdateHD(Node& node) {

    int finalState[N][N] = FINAL_STATE;

    if(node.HD > -1) {
        return;
    }

    // for(int i=0; i<N; i++) {
    //     for(int j=0; j<N; j++) {
    //         cout << node.Data[i][j] << " ";
    //     }
    //     cout << endl;
    // }

    int nodeArr[N*N], finalArr[N*N];
    for(int i=0; i<N; i++) {
        for(int j=0; j<N; j++) {
            int pos = i*N + j;
            // cout << pos << " " << node.Data[i][j] << endl;
            nodeArr[node.Data[i][j]] = pos;
            finalArr[finalState[i][j]] = pos;
            // cout << node.Data[i][j] << " " << pos << endl;
            // int b = getIndexInState(pos, node.Data);
            // int g = getIndexInState(pos, finalState);
            // sum += abs(b % N - g % N);
            // sum += abs(b / N - g / N);
        }
    }

    int HDvalue = 0;
    for(int i=1; i<N*N; i++){
        int b = nodeArr[i];
        int g = finalArr[i];
        HDvalue += abs(b % N - g % N);
        HDvalue += abs(b / N - g / N);
    }
    node.HD = HDvalue;
}

/* 
 * Fill all required values of the struct
 */
CUDA_FUNC void Fill(Node * node, int dt, int hd, int data[N][N], Node * link, int parentID) {
    node->DT = dt;
    node->HD = -1;
    for(int i=0; i<N; i++) {
        for(int j=0; j<N; j++) {
            node->Data[i][j] = data[i][j];
            node->UID[2*(N*i + j)] = data[i][j] + '0';
            node->UID[2*(N*i + j)+1] = ' ';
        }
    }
    node->UID[2*N*N] = '\0';
    node->Link = link;
    node->parentID = parentID;
}

/*
 * This function empty tile in the node.
 * Returns: Add X, Y coordinate of 0 Tile in Array 'arr'
 */
CUDA_FUNC void FindZeros(int data[N][N], int * x, int * y) {
    for(int i=0; i<N; i++) {
        for(int j=0; j<N; j++) {
            if (data[i][j] == 0) {
                *x = i;
                *y = j;
                return;
            }
        }
    }
}

/*
 * DeepcopyData: Copy Data variable in same size array.
 * node: Original Node.
 * copy: Copy data.
 */
CUDA_FUNC void DeepcopyData(Node * node, int copy[N][N]) {
    for (int i=0; i<N; i++) {
        for (int j=0; j<N; j++) {
            copy[i][j] = node->Data[i][j];
        }
    }
}


/* 
 * Find all neighbours of the Node,
 * Next possible state of the node.
 * Case 1: Corners of the Grid.
 * Case 2: When 0 is on the Edge of the grid
 * Case 3: When 0 in anywhere except above above possible cases
 * Return: Add all Neighbours in 2nd Argument
 */
CUDA_FUNC int GetNeighbours(Node * currentNode) {
    Node * neighbours;
    neighbours = (Node *) malloc(sizeof(Node) * 4);
    int x, y, neighbours_count = 0;
    FindZeros(currentNode->Data, &x, &y);
    // cout << "Zero Coordinates\nX: " << x << "\tY: " << y << endl;
    // printf("Printing Current Node in GetNeighbours: HD %d DT %d UID %s Parent %d\n ", currentNode->HD, currentNode->DT, currentNode->UID, currentNode->parentID);
    // Case 1: Corner 1, 2, 3, 4
    if (x == 0 && y == 0) {
        int Temp[N][N];
        DeepcopyData(currentNode, Temp);

        neighbours[0] = {0};
        neighbours[1] = {0};
        neighbours[2] = {0};
        neighbours[3] = {0};

        if (currentNode->parentID != 2) {
            Temp[x][y] = Temp[x][y+1];
            Temp[x][y+1] = 0;
            Node n1;
            Fill(&n1, currentNode->DT+1, -1, Temp, NULL, 4);
            Temp[x][y+1] = Temp[x][y];
            Temp[x][y] = 0;
            neighbours[0] = n1;
            neighbours_count+=1;
        }

        if (currentNode->parentID != 3) {
            Temp[x][y] = Temp[x+1][y];
            Temp[x+1][y] = 0;
            Node n2;
            Fill(&n2, currentNode->DT+1, -1, Temp, NULL, 1);
            Temp[x+1][y] = Temp[x][y];
            Temp[x][y] = 0;
            neighbours[1] = n2;
            neighbours_count+=1;
        }
        // neighbours[2] = {0};
        // neighbours[3] = {0};
        currentNode->Link = neighbours;
        // toString(currentNode);
    }
    else if (x == 0 && y == N-1) {
        // Need to repeat same for rest of the conditions
        int Temp[N][N];
        DeepcopyData(currentNode, Temp);

        neighbours[0] = {0};
        neighbours[1] = {0};
        neighbours[2] = {0};
        neighbours[3] = {0};

        if (currentNode->parentID != 4) {
            Temp[x][y] = Temp[x][y-1];
            Temp[x][y-1] = 0;
            Node n1;
            Fill(&n1, currentNode->DT+1, -1, Temp, NULL, 2);
            Temp[x][y-1] = Temp[x][y];
            Temp[x][y] = 0;
            neighbours[0] = n1;
            neighbours_count+=1;
        }

        if (currentNode->parentID != 3) {
            Temp[x][y] = Temp[x+1][y];
            Temp[x+1][y] = 0;
            Node n2;
            Fill(&n2, currentNode->DT+1, -1, Temp, NULL, 1);
            Temp[x+1][y] = Temp[x][y];
            Temp[x][y] = 0;
            neighbours[1] = n2;
            neighbours_count+=1;
        }
        // neighbours[2] = {0};
        // neighbours[3] = {0};
        currentNode->Link = neighbours;
        // toString(currentNode);
    }
    else if (x == N-1 && y == 0) {
        int Temp[N][N];
        DeepcopyData(currentNode, Temp);

        neighbours[0] = {0};
        neighbours[1] = {0};
        neighbours[2] = {0};
        neighbours[3] = {0};

        if (currentNode->parentID != 2) {
            Temp[x][y] = Temp[x][y+1];
            Temp[x][y+1] = 0;
            Node n1;
            Fill(&n1, currentNode->DT+1, -1, Temp, NULL, 4);
            Temp[x][y+1] = Temp[x][y];
            Temp[x][y] = 0;
            neighbours[0] = n1;
            neighbours_count+=1;
        }

        if (currentNode->parentID != 1) {
            Temp[x][y] = Temp[x-1][y];
            Temp[x-1][y] = 0;
            Node n2;
            Fill(&n2, currentNode->DT+1, -1, Temp, NULL, 3);
            Temp[x-1][y] = Temp[x][y];
            Temp[x][y] = 0;
            neighbours[1] = n2;
            neighbours_count+=1;
        }
        // neighbours[2] = {0};
        // neighbours[3] = {0};
        currentNode->Link = neighbours;
        // toString(currentNode);
    }
    else if (x == N-1 && y == N-1) {
        int Temp[N][N];
        DeepcopyData(currentNode, Temp);

        neighbours[0] = {0};
        neighbours[1] = {0};
        neighbours[2] = {0};
        neighbours[3] = {0};

        if (currentNode->parentID != 4) {
            Temp[x][y] = Temp[x][y-1];
            Temp[x][y-1] = 0;
            Node n1;
            Fill(&n1, currentNode->DT+1, -1, Temp, NULL, 2);
            Temp[x][y-1] = Temp[x][y];
            Temp[x][y] = 0;
            neighbours[0] = n1;
            neighbours_count+=1;
        }

        if (currentNode->parentID != 1) {
            Temp[x][y] = Temp[x-1][y];
            Temp[x-1][y] = 0;
            Node n2;
            Fill(&n2, currentNode->DT+1, -1, Temp, NULL, 3);
            // printf("This is my house %d %d %s\n", n2.DT, n2.HD, n2.UID);
            Temp[x-1][y] = Temp[x][y];
            Temp[x][y] = 0;
            neighbours[1] = n2;
            neighbours_count+=1;
        }
        // neighbours[2] = {0};
        // neighbours[3] = {0};
        currentNode->Link = neighbours;
        // toString(currentNode);
    }
    // Case 2: Edge 1, 2, 3, 4
    else if (x == 0) {
        int Temp[N][N];
        DeepcopyData(currentNode, Temp);

        neighbours[0] = {0};
        neighbours[1] = {0};
        neighbours[2] = {0};
        neighbours[3] = {0};

        if (currentNode->parentID != 2) {
            Temp[x][y] = Temp[x][y+1];
            Temp[x][y+1] = 0;
            Node n1;
            Fill(&n1, currentNode->DT+1, -1, Temp, NULL, 4);
            Temp[x][y+1] = Temp[x][y];
            Temp[x][y] = 0;
            neighbours[0] = n1;
            neighbours_count+=1;
        }

        if (currentNode->parentID != 4) {
            Temp[x][y] = Temp[x][y-1];
            Temp[x][y-1] = 0;
            Node n2;
            Fill(&n2, currentNode->DT+1, -1, Temp, NULL, 2);
            Temp[x][y-1] = Temp[x][y];
            Temp[x][y] = 0;
            neighbours[1] = n2;
            neighbours_count+=1;
        }  

        if (currentNode->parentID != 3) { 
            Temp[x][y] = Temp[x+1][y];
            Temp[x+1][y] = 0;
            Node n3;
            Fill(&n3, currentNode->DT+1, -1, Temp, NULL, 1);
            Temp[x+1][y] = Temp[x][y];
            Temp[x][y] = 0;
            neighbours[2] = n3;
            neighbours_count+=1;
        }
        // neighbours[3] = {0};    
        currentNode->Link = neighbours;
        // toString(currentNode);

    }
    else if (y == 0) {


        int Temp[N][N];
        DeepcopyData(currentNode, Temp);
        neighbours[0] = {0};
        neighbours[1] = {0};
        neighbours[2] = {0};
        neighbours[3] = {0};

        if (currentNode->parentID != 2) {
            Temp[x][y] = Temp[x][y+1];
            Temp[x][y+1] = 0;
            Node n1;
            Fill(&n1, currentNode->DT+1, -1, Temp, NULL, 4);
            Temp[x][y+1] = Temp[x][y];
            Temp[x][y] = 0;
            neighbours[0] = n1;
            neighbours_count+=1;
        }

        if (currentNode->parentID != 1) {
            Temp[x][y] = Temp[x-1][y];
            Temp[x-1][y] = 0;
            Node n2;
            Fill(&n2, currentNode->DT+1, -1, Temp, NULL, 3);
            Temp[x-1][y] = Temp[x][y];
            Temp[x][y] = 0;
            neighbours[1] = n2;
            neighbours_count+=1;
        }

        if (currentNode->parentID != 3) {
            Temp[x][y] = Temp[x+1][y];
            Temp[x+1][y] = 0;
            Node n3;
            Fill(&n3, currentNode->DT+1, -1, Temp, NULL, 1);
            Temp[x+1][y] = Temp[x][y];
            Temp[x][y] = 0;
            neighbours[2] = n3;
            neighbours_count+=1;
        }   
        // neighbours[3] = {0};    
        currentNode->Link = neighbours;
        // toString(currentNode);

    }
    else if (x == N-1) {

        int Temp[N][N];
        DeepcopyData(currentNode, Temp);
        neighbours[0] = {0};
        neighbours[1] = {0};
        neighbours[2] = {0};
        neighbours[3] = {0};

        if (currentNode->parentID !=2) {
            Temp[x][y] = Temp[x][y+1];
            Temp[x][y+1] = 0;
            Node n1;
            Fill(&n1, currentNode->DT+1, -1, Temp, NULL, 4);
            Temp[x][y+1] = Temp[x][y];
            Temp[x][y] = 0;
            neighbours[0] = n1;
            neighbours_count+=1;
        }
       
        if (currentNode->parentID != 4) {
            Temp[x][y] = Temp[x][y-1];
            Temp[x][y-1] = 0;
            Node n2;
            Fill(&n2, currentNode->DT+1, -1, Temp, NULL, 2);
            Temp[x][y-1] = Temp[x][y];
            Temp[x][y] = 0;
            neighbours[1] = n2;
            neighbours_count+=1;
        }
        
        if (currentNode->parentID != 1) {
            Temp[x][y] = Temp[x-1][y];
            Temp[x-1][y] = 0;
            Node n3;
            Fill(&n3, currentNode->DT+1, -1, Temp, NULL, 3);
            Temp[x-1][y] = Temp[x][y];
            Temp[x][y] = 0;
            neighbours[2] = n3;
            neighbours_count+=1;
        }
       
        currentNode->Link = neighbours;
        // toString(currentNode);
    }
    else if (y == N-1) {
        int Temp[N][N];
        DeepcopyData(currentNode, Temp);

        neighbours[0] = {0};
        neighbours[1] = {0};
        neighbours[2] = {0};
        neighbours[3] = {0};

        if (currentNode->parentID != 3) {
            Temp[x][y] = Temp[x+1][y];
            Temp[x+1][y] = 0;
            Node n1;
            Fill(&n1, currentNode->DT+1, -1, Temp, NULL, 1);
            Temp[x+1][y] = Temp[x][y];
            Temp[x][y] = 0;
            neighbours[0] = n1;
            neighbours_count+=1;
        }

        if(currentNode->parentID != 4) {
            Temp[x][y] = Temp[x][y-1];
            Temp[x][y-1] = 0;
            Node n2;
            Fill(&n2, currentNode->DT+1, -1, Temp, NULL, 2);
            Temp[x][y-1] = Temp[x][y];
            Temp[x][y] = 0;
            neighbours[1] = n2;
            neighbours_count+=1;
        }
        
        if(currentNode->parentID != 1) {
            Temp[x][y] = Temp[x-1][y];
            Temp[x-1][y] = 0;
            Node n3;
            Fill(&n3, currentNode->DT+1, -1, Temp, NULL, 3);
            Temp[x-1][y] = Temp[x][y];
            Temp[x][y] = 0;
            neighbours[2] = n3;
            neighbours_count+=1;
        }
        currentNode->Link = neighbours;
        // toString(currentNode);
    }
    // Case 3: Tile 0 Anywhere, except above possible location.
    else {
        int Temp[N][N];
        DeepcopyData(currentNode, Temp);

        neighbours[0] = {0};
        neighbours[1] = {0};
        neighbours[2] = {0};
        neighbours[3] = {0};

        if(currentNode->parentID != 3) {
            Temp[x][y] = Temp[x+1][y];
            Temp[x+1][y] = 0;
            Node n1;
            Fill(&n1, currentNode->DT+1, -1, Temp, NULL, 1);
            Temp[x+1][y] = Temp[x][y];
            Temp[x][y] = 0;
            neighbours[0] = n1;
            neighbours_count+=1;
        }

        if(currentNode->parentID != 4) {
            Temp[x][y] = Temp[x][y-1];
            Temp[x][y-1] = 0;
            Node n2;
            Fill(&n2, currentNode->DT+1, -1, Temp, NULL, 2);
            Temp[x][y-1] = Temp[x][y];
            Temp[x][y] = 0;
            neighbours[1] = n2;
            neighbours_count+=1;
        }

        if(currentNode->parentID != 1) {
            Temp[x][y] = Temp[x-1][y];
            Temp[x-1][y] = 0;
            Node n3;
            Fill(&n3, currentNode->DT+1, -1, Temp, NULL, 3);
            Temp[x-1][y] = Temp[x][y];
            Temp[x][y] = 0;
            neighbours[2] = n3;
            neighbours_count+=1;
        }

        if(currentNode->parentID != 2) {
            Temp[x][y] = Temp[x][y+1];
            Temp[x][y+1] = 0;
            Node n4;
            Fill(&n4, currentNode->DT+1, -1, Temp, NULL, 4);
            Temp[x][y+1] = Temp[x][y];
            Temp[x][y] = 0;
            neighbours[3] = n4;
            neighbours_count+=1;
        }

        currentNode->Link = neighbours;
        // toString(currentNode);
    }

    return neighbours_count;
}

CUDA_FUNC int checkSolution(Node * node, int *FinalState) {
    for (int i=0; i<N; i++) {
        for (int j=0; j<N; j++) {
            if (node->Data[i][j] != FinalState[i*N+j]) {
                return 1;
            }
        }
    }
    return 0;
}

#define K 15
__device__ int continueWork = 1;
__device__ int allQueueEmpty = 0;

__global__ void parAStar(PriorityQueue *pqC, int *FinalState, int startStateHeuristic) {

    int i = blockIdx.x * blockDim.x + threadIdx.x;
    // printf("(%d) ", i);
    int count = 0;

    while(continueWork) {

    	count+=1;
        // printf("This is count of %d value %d\n", i, count);

    	if(i < K) {
            // printf("fdsdfgfd %d\n", pqC[i].getLength());
    		Node val = pqC[i].top();
            // printf("Extracted node in thread: i:%d DT:%d HD:%d UID:%s\n", i, val.DT, val.HD, val.UID);
            if (val.DT != -1) {
                // printf("Extracted node in thread: i:%d DT:%d HD:%d UID:%s\n", i, val.DT, val.HD, val.UID);
                int matchFinal = checkSolution(&val, FinalState);
                if (matchFinal == 0) {
                    printf("Done the solution state\n");
                    continueWork = 0;
                }
                // printf("sdfkhsdlfkjadshlfkajdhlfkjdsh\n");
                // __syncthreads();
                // printf("Generated Neighbours:\n");
    			int count_neighbours = GetNeighbours(&val);
    			for(int j=0; j<4; j++) {
                    UpdateHD(val.Link[j]);
                    if (val.Link[j].DT != 0/* && val.Link[j].DT + val.Link[j].HD <= startStateHeuristic*/) {
                        // printf("UID - %s, HD - %d, DT - %d\n", val.Link[j].UID, val.Link[j].HD, val.Link[j].DT);
                        pqC[(i*4+j)%K].insert(val.Link[j]);
                        // printf("Print neighbour UID %s j %d inK %d HD %d DT %d\n", val.Link[j].UID, j, (i*4+j)%K, val.Link[j].HD, val.Link[j].DT);
                        // printf("True or False %d\n", val.Link[j].parentID == 0);
                  
                    }
                }
                // printf("sdfkhsdlfkjadshlfkajdhlfkjdsh\n");
    		}
            // printf("(%d, %d)\n", i, continueWork);
            __syncthreads();
    	}
        __syncthreads();
    }
}

int emptyPriorityQueue(PriorityQueue pq[K]) {
    for (int i=0; i<K; i++) {
        // printf("Length of queue %d %d\n", pq[i].getLength(), i);
        if (pq[i].getLength() != 0) {
            // pq[i].printHeap();
            return 0;
        }
    }
    return 1;
}

// #define K 320
int main() {



	int Start[N][N] = {
        {4, 5, 1},
        {0, 3, 2},
        {6, 8, 7}
    };

    // int Start[N][N] = {
    //     {12, 1, 10, 2}, 
    //     {7, 11, 4, 14}, 
    //     {5, 0, 9, 15},
    //     {8, 13, 6, 3}
    // };


    //     int Start[N][N] = {
    //     {1, 2, 6, 3}, 
    //     {4, 5, 7, 0}, 
    //     {8, 9, 10, 11},
    //     {12, 13, 14, 15}
    // };


    int FinalState[N][N] = FINAL_STATE;
    Node root, final;
    Fill(&root, 0, 0, Start, NULL, -1);
    Fill(&final, 0, 0, FinalState, NULL, -1);
    UpdateHD(root);
    // printf("root.HD + root.DT = %d\n", root.HD + root.DT);

	int k = K;
	// int blocks = (k / BLOCK_SIZE) + 1;
	// printf("%d %d\n", blocks, BLOCK_SIZE);

	PriorityQueue pq[k];

	// printf("%lu\n", sizeof(PriorityQueue));
	// printf("sdfsddsfsd");
	for(int i=0; i<k; i++) {
		pq[i] = PriorityQueue();
	}

	// printf("%d %d %s\n", root.DT, root.HD, root.UID);
	pq[0].insert(root);

	// int arr[5] = {1,2,3,4,5};
	PriorityQueue *pqC;
    int *FinalState_device;
    int Reached[K];
    for(int i=0; i<K; Reached[i++]=0);
    int *Reached_device;

	hipMalloc((void**)&pqC, sizeof(PriorityQueue)*k);
	
    hipMalloc((void**)&FinalState_device, N*N*sizeof(int));
    
    hipMalloc((void**)&Reached_device, K*(sizeof(int)));
    
    hipMemcpy(pqC, pq, sizeof(PriorityQueue)*k, hipMemcpyHostToDevice);   
    float runningTime = 0;
    // int counter = 0;
    
        hipMemcpy(FinalState_device, FinalState, N*N*sizeof(int), hipMemcpyHostToDevice);
        hipMemcpy(Reached_device, Reached,K*sizeof(int), hipMemcpyHostToDevice);
        
        // printf("HELLO");

        hipEvent_t start_kernel, stop_kernel;
        hipEventCreate(&start_kernel);
        hipEventCreate(&stop_kernel);
        hipEventRecord(start_kernel);
        parAStar<<<1, K>>> (pqC, FinalState_device, root.HD + root.DT);

        hipEventRecord(stop_kernel);
        hipEventSynchronize(stop_kernel);
        float totalTime;
        hipEventElapsedTime(&totalTime, start_kernel, stop_kernel);
        //printf("Tme %f\n", totalTime);
        runningTime += totalTime;
        

    hipMemcpy(pq, pqC, sizeof(PriorityQueue)*k, hipMemcpyDeviceToHost);

    printf("\n\n\n\nFinal Running time %0.4f\n", runningTime);


	return 0;
}