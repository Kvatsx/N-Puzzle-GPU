#include "hip/hip_runtime.h"
#include <iostream>
#include <stdlib.h>
#include <stdio.h>
#include <string.h>
// #include <math.h>
#include <ctime>
#include <bits/stdc++.h> 
#include "Helper.h"
#include <cmath>

using namespace std;

#define BlockSize 32;

__global__ void findZero(int * d_data, unsigned int * d_xy) {
    // printf("%d\t", d_data[threadIdx.x]);
    if (d_data[threadIdx.x] == 0 && threadIdx.x < N*N) {
        d_xy[0] = threadIdx.x;
    }
}

void find_my_neigh(Node& node) {
    int * d_data;
    unsigned int * d_xy;
    hipMalloc((void**) &d_data, sizeof(int)*N*N);
    hipMalloc((void**) &d_xy, sizeof(unsigned int));
    hipMemcpy(d_data, node.Data, sizeof(int)*N*N , hipMemcpyHostToDevice);
    
    findZero<<<1, 32>>>(d_data, d_xy); 
    unsigned int * xy;
    xy = (unsigned int *) malloc(sizeof(unsigned int)); 

    hipMemcpy(xy, d_xy, sizeof(unsigned int), hipMemcpyDeviceToHost);
    // cout << "xy: " << xy[0] << endl;
    int x = (int) floor(xy[0] / N);
    int y = (int)xy[0] - x*N;
    // cout << "x: " << x << " y: " << y << endl;
    
    // Case 1: Corner 1, 2, 3, 4
    if (x == 0 && y == 0) {
        int Temp[N][N];
        DeepcopyData(currentNode, Temp);
        Temp[x][y] = Temp[x][y+1];
        Temp[x][y+1] = 0;
        Node n1;
        Fill(&n1, currentNode->DT+1, -1, Temp, NULL);
        Temp[x][y+1] = Temp[x][y];
        Temp[x][y] = 0;
        neighbours[0] = n1;

        Temp[x][y] = Temp[x+1][y];
        Temp[x+1][y] = 0;
        Node n2;
        Fill(&n2, currentNode->DT+1, -1, Temp, NULL);
        Temp[x+1][y] = Temp[x][y];
        Temp[x][y] = 0;
        neighbours[1] = n2;
        neighbours[2] = {0};
        neighbours[3] = {0};
        currentNode->Link = neighbours;
        // toString(currentNode);
    }
    else if (x == 0 && y == N-1) {
        // Need to repeat same for rest of the conditions
        int Temp[N][N];
        DeepcopyData(currentNode, Temp);
        Temp[x][y] = Temp[x][y-1];
        Temp[x][y-1] = 0;
        Node n1;
        Fill(&n1, currentNode->DT+1, -1, Temp, NULL);
        Temp[x][y-1] = Temp[x][y];
        Temp[x][y] = 0;
        neighbours[0] = n1;

        Temp[x][y] = Temp[x+1][y];
        Temp[x+1][y] = 0;
        Node n2;
        Fill(&n2, currentNode->DT+1, -1, Temp, NULL);
        Temp[x+1][y] = Temp[x][y];
        Temp[x][y] = 0;
        neighbours[1] = n2;
        neighbours[2] = {0};
        neighbours[3] = {0};
        currentNode->Link = neighbours;
        // toString(currentNode);
    }
    else if (x == N-1 && y == 0) {
        int Temp[N][N];
        DeepcopyData(currentNode, Temp);
        Temp[x][y] = Temp[x][y+1];
        Temp[x][y+1] = 0;
        Node n1;
        Fill(&n1, currentNode->DT+1, -1, Temp, NULL);
        Temp[x][y+1] = Temp[x][y];
        Temp[x][y] = 0;
        neighbours[0] = n1;

        Temp[x][y] = Temp[x-1][y];
        Temp[x-1][y] = 0;
        Node n2;
        Fill(&n2, currentNode->DT+1, -1, Temp, NULL);
        Temp[x-1][y] = Temp[x][y];
        Temp[x][y] = 0;
        neighbours[1] = n2;
        neighbours[2] = {0};
        neighbours[3] = {0};
        currentNode->Link = neighbours;
        // toString(currentNode);
    }
    else if (x == N-1 && y == N-1) {
        int Temp[N][N];
        DeepcopyData(currentNode, Temp);
        Temp[x][y] = Temp[x][y-1];
        Temp[x][y-1] = 0;
        Node n1;
        Fill(&n1, currentNode->DT+1, -1, Temp, NULL);
        Temp[x][y-1] = Temp[x][y];
        Temp[x][y] = 0;
        neighbours[0] = n1;

        Temp[x][y] = Temp[x-1][y];
        Temp[x-1][y] = 0;
        Node n2;
        Fill(&n2, currentNode->DT+1, -1, Temp, NULL);
        Temp[x-1][y] = Temp[x][y];
        Temp[x][y] = 0;
        neighbours[1] = n2;
        neighbours[2] = {0};
        neighbours[3] = {0};
        currentNode->Link = neighbours;
        // toString(currentNode);
    }
    // Case 2: Edge 1, 2, 3, 4
    else if (x == 0) {
        int Temp[N][N];
        DeepcopyData(currentNode, Temp);
        Temp[x][y] = Temp[x][y+1];
        Temp[x][y+1] = 0;
        Node n1;
        Fill(&n1, currentNode->DT+1, -1, Temp, NULL);
        Temp[x][y+1] = Temp[x][y];
        Temp[x][y] = 0;
        neighbours[0] = n1;

        Temp[x][y] = Temp[x][y-1];
        Temp[x][y-1] = 0;
        Node n2;
        Fill(&n2, currentNode->DT+1, -1, Temp, NULL);
        Temp[x][y-1] = Temp[x][y];
        Temp[x][y] = 0;
        neighbours[1] = n2;

        Temp[x][y] = Temp[x+1][y];
        Temp[x+1][y] = 0;
        Node n3;
        Fill(&n3, currentNode->DT+1, -1, Temp, NULL);
        Temp[x+1][y] = Temp[x][y];
        Temp[x][y] = 0;
        neighbours[2] = n3;
        neighbours[3] = {0};    
        currentNode->Link = neighbours;
        // toString(currentNode);

    }
    else if (y == 0) {
        int Temp[N][N];
        DeepcopyData(currentNode, Temp);
        Temp[x][y] = Temp[x][y+1];
        Temp[x][y+1] = 0;
        Node n1;
        Fill(&n1, currentNode->DT+1, -1, Temp, NULL);
        Temp[x][y+1] = Temp[x][y];
        Temp[x][y] = 0;
        neighbours[0] = n1;

        Temp[x][y] = Temp[x-1][y];
        Temp[x-1][y] = 0;
        Node n2;
        Fill(&n2, currentNode->DT+1, -1, Temp, NULL);
        Temp[x-1][y] = Temp[x][y];
        Temp[x][y] = 0;
        neighbours[1] = n2;

        Temp[x][y] = Temp[x+1][y];
        Temp[x+1][y] = 0;
        Node n3;
        Fill(&n3, currentNode->DT+1, -1, Temp, NULL);
        Temp[x+1][y] = Temp[x][y];
        Temp[x][y] = 0;
        neighbours[2] = n3;
        neighbours[3] = {0};    
        currentNode->Link = neighbours;
        // toString(currentNode);

    }
    else if (x == N-1) {
        int Temp[N][N];
        DeepcopyData(currentNode, Temp);
        Temp[x][y] = Temp[x][y+1];
        Temp[x][y+1] = 0;
        Node n1;
        Fill(&n1, currentNode->DT+1, -1, Temp, NULL);
        Temp[x][y+1] = Temp[x][y];
        Temp[x][y] = 0;
        neighbours[0] = n1;

        Temp[x][y] = Temp[x][y-1];
        Temp[x][y-1] = 0;
        Node n2;
        Fill(&n2, currentNode->DT+1, -1, Temp, NULL);
        Temp[x][y-1] = Temp[x][y];
        Temp[x][y] = 0;
        neighbours[1] = n2;

        Temp[x][y] = Temp[x-1][y];
        Temp[x-1][y] = 0;
        Node n3;
        Fill(&n3, currentNode->DT+1, -1, Temp, NULL);
        Temp[x-1][y] = Temp[x][y];
        Temp[x][y] = 0;
        neighbours[2] = n3;
        neighbours[3] = {0};
        currentNode->Link = neighbours;
        // toString(currentNode);
    }
    else if (y == N-1) {
        int Temp[N][N];
        DeepcopyData(currentNode, Temp);
        Temp[x][y] = Temp[x+1][y];
        Temp[x+1][y] = 0;
        Node n1;
        Fill(&n1, currentNode->DT+1, -1, Temp, NULL);
        Temp[x+1][y] = Temp[x][y];
        Temp[x][y] = 0;
        neighbours[0] = n1;

        Temp[x][y] = Temp[x][y-1];
        Temp[x][y-1] = 0;
        Node n2;
        Fill(&n2, currentNode->DT+1, -1, Temp, NULL);
        Temp[x][y-1] = Temp[x][y];
        Temp[x][y] = 0;
        neighbours[1] = n2;

        Temp[x][y] = Temp[x-1][y];
        Temp[x-1][y] = 0;
        Node n3;
        Fill(&n3, currentNode->DT+1, -1, Temp, NULL);
        Temp[x-1][y] = Temp[x][y];
        Temp[x][y] = 0;
        neighbours[2] = n3;
        neighbours[3] = {0};
        currentNode->Link = neighbours;
        // toString(currentNode);
    }
    // Case 3: Tile 0 Anywhere, except above possible location.
    else {
        int Temp[N][N];
        DeepcopyData(currentNode, Temp);
        Temp[x][y] = Temp[x+1][y];
        Temp[x+1][y] = 0;
        Node n1;
        Fill(&n1, currentNode->DT+1, -1, Temp, NULL);
        Temp[x+1][y] = Temp[x][y];
        Temp[x][y] = 0;
        neighbours[0] = n1;

        Temp[x][y] = Temp[x][y-1];
        Temp[x][y-1] = 0;
        Node n2;
        Fill(&n2, currentNode->DT+1, -1, Temp, NULL);
        Temp[x][y-1] = Temp[x][y];
        Temp[x][y] = 0;
        neighbours[1] = n2;

        Temp[x][y] = Temp[x-1][y];
        Temp[x-1][y] = 0;
        Node n3;
        Fill(&n3, currentNode->DT+1, -1, Temp, NULL);
        Temp[x-1][y] = Temp[x][y];
        Temp[x][y] = 0;
        neighbours[2] = n3;

        Temp[x][y] = Temp[x][y+1];
        Temp[x][y+1] = 0;
        Node n4;
        Fill(&n4, currentNode->DT+1, -1, Temp, NULL);
        Temp[x][y+1] = Temp[x][y];
        Temp[x][y] = 0;
        neighbours[3] = n4;
        currentNode->Link = neighbours;
        // toString(currentNode);
    }

    return;
}

// class GetNeighbours {
// public:

    
// };

int main() {
    int Start[][N] = {
        {3, 1, 2, 5}, {4, 0, 6, 7}, {8, 9, 10, 11}, {12, 13, 14, 15}
    };
    Node root;
    Fill(&root, 0, 0, Start, NULL);
    // GetNeighbours gn;
    find_my_neigh(root);
}
